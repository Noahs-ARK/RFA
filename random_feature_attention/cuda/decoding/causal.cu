#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>
#include <vector>
#include <stdio.h>
#include "utils.cu"


__forceinline__ __device__
void read_sz1(
    const __half * __restrict__ s_local,
    const __half * __restrict__ z_local,
    __half2 s_val[HALF2_PER_THREAD],
    __half2 z_val[HALF2_PER_THREAD]) {
    #pragma unroll
    for (int j = 0; j < INT4_PER_THREAD; ++ j) {
        *((int4 *) s_val + j) = *((int4*) s_local + j);
    }

    #pragma unroll
    for (int j = 0; j < INT4_PER_THREAD; ++ j) {
        *((int4 *) z_val + j) = *((int4*) z_local + j);
    }
}


__forceinline__ __device__
void write_sz1(
    const __half * __restrict__ s_local,
    const __half * __restrict__ z_local,
    __half2 s_val[HALF2_PER_THREAD],
    __half2 z_val[HALF2_PER_THREAD]) {
    #pragma unroll
    for (int j = 0; j < INT4_PER_THREAD; ++ j) {
        *((int4 *) s_local + j) = *((int4*) s_val + j);
    }

    if (threadIdx.y == 0) {
        #pragma unroll
        for (int j = 0; j < INT4_PER_THREAD; ++ j) {
            *((int4 *) z_local + j) = *((int4*) z_val + j);
        }
    }
}


__device__
void causal_rfa_step(
        const __half * __restrict__ q_local,
        const __half * __restrict__ k_local,
        const __half * __restrict__ v_local,
        __half * __restrict__ attn_local,
        int num_threads_per_head_dim,
        __half2 s_val[HALF2_PER_THREAD], 
        __half2 z_val[HALF2_PER_THREAD]) {
    __half2 q_val[HALF2_PER_THREAD] = { __float2half2_rn(0.f)};
    __half2 k_val[HALF2_PER_THREAD] = { __float2half2_rn(0.f)};
    #pragma unroll
    for (int j = 0; j < INT4_PER_THREAD; ++ j) {
        *((int4 *) q_val + j) = *((int4*) q_local + j);
    }
    #pragma unroll
    for (int j = 0; j < INT4_PER_THREAD; ++ j) {
        *((int4 *) k_val + j) = *((int4*) k_local + j);
    }

    __half v_half = *v_local;
    __half2 v_val = __half2half2(v_half);
    __half2 qs = __float2half2_rn(0.f);
    __half2 qz = __float2half2_rn(0.f);
    #pragma unroll 
    for (int i = 0;i < HALF2_PER_THREAD; ++ i) {
        s_val[i] = __hfma2(v_val, k_val[i], s_val[i]);
        qs = __hfma2(s_val[i], q_val[i], qs);
        
        z_val[i] = __hadd2(k_val[i], z_val[i]);
        qz = __hfma2(z_val[i], q_val[i], qz);
    }
    #pragma unroll 
    for (int offset = num_threads_per_head_dim >>= 1;
         offset > 0; 
         offset >>= 1) {
        qz =  __hadd2(qz, __shfl_down_sync(FULL_MASK, qz, offset));
        qs =  __hadd2(qs, __shfl_down_sync(FULL_MASK, qs, offset));
    }
    __half qs_half = __hadd(qs.x, qs.y);
    __half qz_half = __hadd(qz.x, qz.y);
    qz_half = clamp_eps(qz_half);
    
    if (threadIdx.x == 0) {
        *attn_local = __hdiv(qs_half, qz_half);
    }
}


__global__ 
void causal_rfa(
        const __half * __restrict__ q,
        const __half * __restrict__ k,
        const __half * __restrict__ v,
        __half * __restrict__ s,
        __half * __restrict__ z,
        __half * __restrict__ attn,
        int head_dim, 
        int proj_dim,
        int num_threads_per_head_dim,
        int num_head_dim_per_block,
        int num_blocks_per_batch) {
    /*
    Args:
        q: [tgt_len, bsz, proj_dim]
        k: [tgt_len, bsz, proj_dim]
        v: [tgt_len, bsz, head_dim]
        s: [bsz, head_dim, proj_dim]
        z: [bsz, proj_dim]
        attn: [tgt_len, bsz, head_dim]
    */
    const int batch_id = blockIdx.x / num_blocks_per_batch;
    const int proj_dim_offset = threadIdx.x * DIM_PER_THREAD;
    const int head_dim_id \
        = (blockIdx.x % num_blocks_per_batch) * num_head_dim_per_block + threadIdx.y;

    const __half * __restrict__ q_local = q + batch_id * proj_dim + proj_dim_offset;
    const __half * __restrict__ k_local = k + batch_id * proj_dim  + proj_dim_offset;
    const __half * __restrict__ v_local = v + batch_id * head_dim + head_dim_id;
    
    __half * __restrict__ s_local \
            = s + batch_id * head_dim * proj_dim \
                + head_dim_id * proj_dim + proj_dim_offset;
    __half * __restrict__ z_local = z + batch_id * proj_dim + proj_dim_offset;
    __half * __restrict__ attn_local = attn + batch_id * head_dim + head_dim_id;

    __half2 s_val[HALF2_PER_THREAD] = {__float2half2_rn(0.f)};
    __half2 z_val[HALF2_PER_THREAD] = {__float2half2_rn(0.f)};
    
    read_sz1(s_local, z_local, s_val, z_val);
    causal_rfa_step(
        q_local, k_local, v_local,
        attn_local,
        num_threads_per_head_dim, 
        s_val, z_val
    );
    write_sz1(s_local, z_local, s_val, z_val);
}


std::vector<Tensor> CausalRFA(
        Tensor const& q,
        Tensor const& k,
        Tensor const& v,
        Tensor & s,
        Tensor & z) {
    /*
    Args:
        q: [tgt_len, bsz, proj_dim]
        k: [tgt_len, bsz, proj_dim]
        v: [tgt_len, bsz, head_dim]
        s: [bsz, head_dim, proj_dim]
        z: [bsz, proj_dim]
        
    Return:
        attn: [tgt_len, bsz, head_dim]
        s: [bsz, head_dim, proj_dim]
        z: [bsz, proj_dim]
    */
    // column major
    const int bsz = q.size(1);
    const int proj_dim = q.size(2);
    const int head_dim = v.size(2);

    auto act_options  = q.options().requires_grad(false);
    Tensor attn = torch::zeros({1, bsz, head_dim}, act_options);
    
    // num threads per head_dim;
    int num_threads_per_head_dim = proj_dim / DIM_PER_THREAD;
    const int num_head_dim_per_block = min(
        head_dim, NUM_THREADS_PER_BLOCK / num_threads_per_head_dim); 
    const int num_blocks_per_batch = max(1, head_dim / num_head_dim_per_block);

    dim3 dim_grid(bsz * num_blocks_per_batch);
    // [x, y]
    dim3 dim_block(num_threads_per_head_dim, num_head_dim_per_block);
    causal_rfa <<<dim_grid, dim_block>>>(
            static_cast<const __half *> (q.data_ptr()), 
            static_cast<const __half *> (k.data_ptr()), 
            static_cast<const __half *> (v.data_ptr()), 
            static_cast<__half *> (s.data_ptr()), 
            static_cast<__half *> (z.data_ptr()), 
            static_cast<__half *> (attn.data_ptr()), 
            head_dim, 
            proj_dim,
            num_threads_per_head_dim,
            num_head_dim_per_block,
            num_blocks_per_batch
    );
 
    return {attn, s, z};
}
